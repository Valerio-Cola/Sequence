#include "hip/hip_runtime.h"
/*
 * Exact genetic sequence alignment
 * (Using brute force)
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2023/2024
 *
 * v1.3 
 *
 * (c) 2024, Arturo Gonzalez-Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<limits.h>
#include<sys/time.h>
#include<mpi.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/* Example of macros for error checking in CUDA */
#define CUDA_CHECK_FUNCTION( call )	{ hipError_t check = call; if ( check != hipSuccess ) fprintf(stderr, "CUDA Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }
#define CUDA_CHECK_KERNEL( )	{ hipError_t check = hipGetLastError(); if ( check != hipSuccess ) fprintf(stderr, "CUDA Kernel Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }

/* Arbitrary value to indicate that no matches are found */
#define	NOT_FOUND	-1

/* Arbitrary value to restrict the checksums period */
#define CHECKSUM_MAX	65535


/* 
 * Utils: Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Utils: Random generator
 */
#include "rng.c"


/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
/* ADD KERNELS AND OTHER FUNCTIONS HERE */

/*
 * Function: Increment the number of pattern matches on the sequence positions
 * 	This function can be changed and/or optimized by the students
 */
__device__ void increment_matches( int pat, unsigned long *pat_found, unsigned long *pat_length, int *seq_matches ) {
	unsigned long ind;
	//__syncthreads();	
	for( ind=0; ind<pat_length[pat]; ind++) {
			atomicAdd(&seq_matches[ pat_found[pat] + ind ], 1);
			//seq_matches[ pat_found[pat] + ind ] ++;
	}
	//__syncthreads();
}

__global__ void sequencer(unsigned long *g_seq_length, int *g_pat_number, char *g_sequence, unsigned long *d_pat_length, char **d_pattern, int *g_seq_matches, int *g_pat_matches, unsigned long *g_pat_found, int *g_my_first_pattern) { 
    unsigned long start;
    int pat;
    unsigned long lind;
    /* Se vogliamo fare che ogni thread ha una sola sequenza da cercare
            Questo primo for è inutile
            il nostro thread int i = blockIdx.x * blockDim.x + threadIdx.x; (il prof ha fatto una cosa simile nella moltiplicazione tra vettori)
            verifichiamo che intanto sia uno di quelli che deve lavorare indipendentemente dal blocco:  i < g_pat_number
            e poi utilizziamo il suo indice come variabile pat  quindi pat = i nella dichiarazione.
            Io ho intanto pensato a questa implementazione se vuoi fare in unaltro modo non cancellare questi commenti.

            noi possiamo trovare il (o i) pattern da cercare basandoci sull'indice del thread
            se facciamo che ogni thread cerca solo un pattern dobbiamo organizzare i thread nel blocco in un certo modo
    */

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("Total patterns to find: %d\n", *g_pat_number);
    if (tid < *g_pat_number) {
        //printf("Thread %d is working\n", tid);
        // esegui il lavoro solo se l'ID del thread è valido
        pat = tid;
        /* 5.1. For each posible starting position */
        for( start=*g_my_first_pattern; start <= *g_seq_length - d_pat_length[pat]; start++) {
            /* 5.1.1. For each pattern element */
            for( lind=0; lind<d_pat_length[pat]; lind++) {

                /* Stop this test when different nucleotids are found */
                if ( g_sequence[start + lind] != d_pattern[pat][lind] ) break;
            }
            /* 5.1.2. Check if the loop ended with a match */
            if ( lind == d_pat_length[pat] ) {
                //printf("Pattern %d found at position %lu Tid: %d lind: %lu pat_lenght: %lu\n", pat, start, tid, lind, d_pat_length[pat]);
                // qua ho tolto il & perché era un errore di indirizzamento
                atomicAdd(g_pat_matches, 1);
                //printf("Thread %d: Total pattern matches: %d\n", tid, atomicAdd(g_pat_matches, 0));
                // qua invece ho castato le variabili in unsigned long long non so perché prima non andasse bene
                atomicExch((unsigned long long*)&g_pat_found[pat], (unsigned long long)start);
                break;
            }
        }

        /* 5.2. Pattern found */
        if ( g_pat_found[pat] != (unsigned long)NOT_FOUND ) {
            /* 4.2.1. Increment the number of pattern matches on the sequence positions */
            increment_matches( pat, g_pat_found, d_pat_length, g_seq_matches );
        }
		//__syncthreads();
	}

}

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

/*
 * Function: Allocate new patttern
 */
char *pattern_allocate( rng_t *random, unsigned long pat_rng_length_mean, unsigned long pat_rng_length_dev, unsigned long seq_length, unsigned long *new_length ) {

	/* Random length */
	unsigned long length = (unsigned long)rng_next_normal( random, (double)pat_rng_length_mean, (double)pat_rng_length_dev );
	if ( length > seq_length ) length = seq_length;
	if ( length <= 0 ) length = 1;

	/* Allocate pattern */
	char *pattern = (char *)malloc( sizeof(char) * length );
	if ( pattern == NULL ) {
		fprintf(stderr,"\n-- Error allocating a pattern of size: %lu\n", length );
		exit( EXIT_FAILURE );
	}

	/* Return results */
	*new_length = length;
	return pattern;
}

/*
 * Function: Fill random sequence or pattern
 */
void generate_rng_sequence( rng_t *random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length) {
	unsigned long ind; 
	for( ind=0; ind<length; ind++ ) {
		double prob = rng_next( random );
		if( prob < prob_G ) seq[ind] = 'G';
		else if( prob < prob_C ) seq[ind] = 'C';
		else if( prob < prob_A ) seq[ind] = 'A';
		else seq[ind] = 'T';
	}
}

/*
 * Function: Copy a sample of the sequence
 */
void copy_sample_sequence( rng_t *random, char *sequence, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Copy sample */
	unsigned long ind; 
	for( ind=0; ind<length; ind++ )
		pattern[ind] = sequence[ind+location];
}

/*
 * Function: Regenerate a sample of the sequence
 */
void generate_sample_sequence( rng_t *random, rng_t random_seq, float prob_G, float prob_C, float prob_A, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length ) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Regenerate sample */
	rng_t local_random = random_seq;
	rng_skip( &local_random, location );
	generate_rng_sequence( &local_random, prob_G, prob_C, prob_A, pattern, length);
}


/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<seq_length> <prob_G> <prob_C> <prob_A> <pat_rng_num> <pat_rng_length_mean> <pat_rng_length_dev> <pat_samples_num> <pat_samp_length_mean> <pat_samp_length_dev> <pat_samp_loc_mean> <pat_samp_loc_dev> <pat_samp_mix:B[efore]|A[fter]|M[ixed]> <long_seed>\n");
	fprintf(stderr,"\n");
}



/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	/* 0. Default output and error without buffering, forces to write immediately */
	setbuf(stdout, NULL);
	setbuf(stderr, NULL);

	/* 1. Read scenary arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc < 15) {
		fprintf(stderr, "\n-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	unsigned long seq_length = atol( argv[1] );
	float prob_G = atof( argv[2] );
	float prob_C = atof( argv[3] );
	float prob_A = atof( argv[4] );
	if ( prob_G + prob_C + prob_A > 1 ) {
		fprintf(stderr, "\n-- Error: The sum of G,C,A,T nucleotid probabilities cannot be higher than 1\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}
	prob_C += prob_G;
	prob_A += prob_C;

	int pat_rng_num = atoi( argv[5] );
	unsigned long pat_rng_length_mean = atol( argv[6] );
	unsigned long pat_rng_length_dev = atol( argv[7] );
	
	int pat_samp_num = atoi( argv[8] );
	unsigned long pat_samp_length_mean = atol( argv[9] );
	unsigned long pat_samp_length_dev = atol( argv[10] );
	unsigned long pat_samp_loc_mean = atol( argv[11] );
	unsigned long pat_samp_loc_dev = atol( argv[12] );

	char pat_samp_mix = argv[13][0];
	if ( pat_samp_mix != 'B' && pat_samp_mix != 'A' && pat_samp_mix != 'M' ) {
		fprintf(stderr, "\n-- Error: Incorrect first character of pat_samp_mix: %c\n\n", pat_samp_mix);
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	unsigned long seed = atol( argv[14] );

#ifdef DEBUG
	/* DEBUG: Print arguments */
	printf("\nArguments: seq_length=%lu\n", seq_length );
	printf("Arguments: Accumulated probabilitiy G=%f, C=%f, A=%f, T=1\n", prob_G, prob_C, prob_A );
	printf("Arguments: Random patterns number=%d, length_mean=%lu, length_dev=%lu\n", pat_rng_num, pat_rng_length_mean, pat_rng_length_dev );
	printf("Arguments: Sample patterns number=%d, length_mean=%lu, length_dev=%lu, loc_mean=%lu, loc_dev=%lu\n", pat_samp_num, pat_samp_length_mean, pat_samp_length_dev, pat_samp_loc_mean, pat_samp_loc_dev );
	printf("Arguments: Type of mix: %c, Random seed: %lu\n", pat_samp_mix, seed );
	printf("\n");
#endif // DEBUG

        CUDA_CHECK_FUNCTION( hipSetDevice(0) );

	/* 2. Initialize data structures */
	/* 2.1. Skip allocate and fill sequence */
	rng_t random = rng_new( seed );
	rng_skip( &random, seq_length );

	/* 2.2. Allocate and fill patterns */
	/* 2.2.1 Allocate main structures */
	int pat_number = pat_rng_num + pat_samp_num;
	unsigned long *pat_length = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	char **pattern = (char **)malloc( sizeof(char*) * pat_number );
	if ( pattern == NULL || pat_length == NULL ) {
		fprintf(stderr,"\n-- Error allocating the basic patterns structures for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}

	/* 2.2.2 Allocate and initialize ancillary structure for pattern types */
	int ind;
	unsigned long lind;
	#define PAT_TYPE_NONE	0
	#define PAT_TYPE_RNG	1
	#define PAT_TYPE_SAMP	2
	char *pat_type = (char *)malloc( sizeof(char) * pat_number );
	if ( pat_type == NULL ) {
		fprintf(stderr,"\n-- Error allocating ancillary structure for pattern of size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_NONE;

	/* 2.2.3 Fill up pattern types using the chosen mode */
	switch( pat_samp_mix ) {
	case 'A':
		for( ind=0; ind<pat_rng_num; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		break;
	case 'B':
		for( ind=0; ind<pat_samp_num; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		break;
	default:
		if ( pat_rng_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		}
		else if ( pat_samp_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		}
		else if ( pat_rng_num < pat_samp_num ) {
			int interval = pat_number / pat_rng_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_RNG;
				else pat_type[ind] = PAT_TYPE_SAMP;
		}
		else {
			int interval = pat_number / pat_samp_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_SAMP;
				else pat_type[ind] = PAT_TYPE_RNG;
		}
	}

	/* 2.2.4 Generate the patterns */
	for( ind=0; ind<pat_number; ind++ ) {
		if ( pat_type[ind] == PAT_TYPE_RNG ) {
			pattern[ind] = pattern_allocate( &random, pat_rng_length_mean, pat_rng_length_dev, seq_length, &pat_length[ind] );
			generate_rng_sequence( &random, prob_G, prob_C, prob_A, pattern[ind], pat_length[ind] );
		}
		else if ( pat_type[ind] == PAT_TYPE_SAMP ) {
			pattern[ind] = pattern_allocate( &random, pat_samp_length_mean, pat_samp_length_dev, seq_length, &pat_length[ind] );
#define REGENERATE_SAMPLE_PATTERNS
#ifdef REGENERATE_SAMPLE_PATTERNS
			rng_t random_seq_orig = rng_new( seed );
			generate_sample_sequence( &random, random_seq_orig, prob_G, prob_C, prob_A, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#else
			copy_sample_sequence( &random, sequence, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#endif
		}
		else {
			fprintf(stderr,"\n-- Error internal: Paranoic check! A pattern without type at position %d\n", ind );
			exit( EXIT_FAILURE );
		}
	}
	free( pat_type );

	/* Allocate and move the patterns to the GPU */
	unsigned long *d_pat_length;
	char **d_pattern;
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pat_length, sizeof(unsigned long) * pat_number ) );
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pattern, sizeof(char *) * pat_number ) );

	char **d_pattern_in_host = (char **)malloc( sizeof(char*) * pat_number );
	if ( d_pattern_in_host == NULL ) {
		fprintf(stderr,"\n-- Error allocating the patterns structures replicated in the host for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) {
		CUDA_CHECK_FUNCTION( hipMalloc( &(d_pattern_in_host[ind]), sizeof(char *) * pat_length[ind] ) );
        	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern_in_host[ind], pattern[ind], pat_length[ind] * sizeof(char), hipMemcpyHostToDevice ) );
	}
	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern, d_pattern_in_host, pat_number * sizeof(char *), hipMemcpyHostToDevice ) );

	/* Avoid the usage of arguments to take strategic decisions
	 * In a real case the user only has the patterns and sequence data to analize
	 */
	argc = 0;
	argv = NULL;
	pat_rng_num = 0;
	pat_rng_length_mean = 0;
	pat_rng_length_dev = 0;
	pat_samp_num = 0;
	pat_samp_length_mean = 0;
	pat_samp_length_dev = 0;
	pat_samp_loc_mean = 0;
	pat_samp_loc_dev = 0;
	pat_samp_mix = '0';

	/* 2.3. Other result data and structures */
	int pat_matches = 0;

	/* 2.3.1. Other results related to patterns */
	unsigned long *pat_found;
	pat_found = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	if ( pat_found == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux pattern structure for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	
	/* 3. Start global timer */
        CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */

    // Inizializzazione MPI
    MPI_Init( &argc, &argv );

	int rank;
	MPI_Comm_rank( MPI_COMM_WORLD, &rank );

    int size;
	MPI_Comm_size( MPI_COMM_WORLD, &size );

	// Idealmente 1 rank per GPU
    // Ognuno si prende tot pattern da cercare
	int my_pat_number = pat_number/size;
	int resto = pat_number % size;

	// Si tiene traccia del primo e dell'ultimo pattern
	int my_first_pattern = rank * my_pat_number;
	
	if(rank == size - 1){
		my_pat_number += resto;
	}

	MPI_Barrier( MPI_COMM_WORLD );

	/* 2.1. Allocate and fill sequence */
	char *sequence = (char *)malloc( sizeof(char) * seq_length );
	if ( sequence == NULL ) {
		fprintf(stderr,"\n-- Error allocating the sequence for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

    if (rank == 0) {
	    random = rng_new( seed );
	    generate_rng_sequence( &random, prob_G, prob_C, prob_A, sequence, seq_length);
    }

    MPI_Bcast(sequence, seq_length, MPI_CHAR, 0, MPI_COMM_WORLD);

	MPI_Barrier( MPI_COMM_WORLD );

#ifdef DEBUG
	/* DEBUG: Print sequence and patterns */
	printf("-----------------\n");
	printf("Sequence: ");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( "%c", sequence[lind] );
	printf("\n-----------------\n");
	printf("Patterns: %d ( rng: %d, samples: %d )\n", pat_number, pat_rng_num, pat_samp_num );
	int debug_pat;
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( "Pat[%d]: ", debug_pat );
		for( lind=0; lind<pat_length[debug_pat]; lind++ ) 
			printf( "%c", pattern[debug_pat][lind] );
		printf("\n");
	}
	printf("-----------------\n\n");
#endif // DEBUG

	/* 2.3.2. Other results related to the main sequence */
	int *seq_matches;
	seq_matches = (int *)malloc( sizeof(int) * seq_length );
	if ( seq_matches == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux sequence structures for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	/* 4. Initialize ancillary structures */
	for( ind=0; ind<pat_number; ind++) {
		pat_found[ind] = (unsigned long)NOT_FOUND;
	}

	for( lind=0; lind<seq_length; lind++) {
		seq_matches[lind] = NOT_FOUND;
	}
    
	unsigned long *my_pat_found;
	my_pat_found = (unsigned long *)malloc( sizeof(unsigned long) * my_pat_number );
	if ( my_pat_found == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux pattern structure for size: %d\n", my_pat_number );
		exit( EXIT_FAILURE );
	}

	for( ind=0; ind<my_pat_number; ind++) {
		my_pat_found[ind] = (unsigned long)NOT_FOUND;
	}

	MPI_Barrier( MPI_COMM_WORLD );

    /*
    Vorrei vedere se si può implementare MPI non soltanto per rendere possibile l'utilizzo di più GPU ma anche
    per ovviare al problema della memoria non sufficiente per pattern molto grandi, vedo già che questo errore
    non possiamo direttamente risolverlo così perché l'allocazione di tutti i pattern avviene in un area del codice non modificabile (righe 361,362)
    */

    // Comando usato per compilare nel cluster:
    // nvcc -O3 -Xcompiler -Wall -arch=sm_75 align_mpi.cu rng.c -o align_mpi -I/usr/lib/x86_64-linux-gnu/openmpi/include/openmpi -I/usr/lib/x86_64-linux-gnu/openmpi/include -lm -lmpi

	// nvcc -arch=sm_75 align_mpi.cu -o align_mpi -I/usr/lib/x86_64-linux-gnu/openmpi/include -L/usr/lib/x86_64-linux-gnu/openmpi/lib -lmpi


	// Variabili che non verranno modificate le sposto nella constant memory
	// NOTA: d_pattern e d_pat_lenght gia allocati in GPU
	unsigned long *g_seq_length;
	int *g_pat_number;
	char *g_sequence;

	/* 
	alla fine ho lasciato tutto in global memory per due motivi: perché la memoria globale è cached 
	e poi perché pattern e pat_length sono già in GPU allocati in global nella sezione di codice che non si può modificare,
	quindi lo stesso ragionamento si applica anche sequence
	inoltre con la clausola __constant__ vai ad usare già la shared memory (secondo le slide del prof)
	*/

	//hipMemcpyToSymbol(HIP_SYMBOL(g_seq_length), &seq_length, sizeof(unsigned long));
	//hipMemcpyToSymbol(HIP_SYMBOL(g_sequence), sequence, seq_length * sizeof(char));
	//hipMemcpyToSymbol(HIP_SYMBOL(g_pat_number), &pat_number, sizeof(int));


	// Necessariamente nella globale della GPU nel caso ci siano più blocchi che devono modificare
	int *g_seq_matches;
	int *g_pat_matches;
	unsigned long *g_pat_found;

	int *g_my_first_pattern;


	hipMalloc(&g_seq_matches, seq_length * sizeof(int));
	hipMalloc(&g_pat_matches, sizeof(int));
	hipMalloc(&g_pat_found, my_pat_number * sizeof(unsigned long));
	hipMalloc(&g_seq_length, sizeof(unsigned long));
	hipMalloc(&g_pat_number, sizeof(int));
	hipMalloc(&g_sequence, seq_length * sizeof(char));
	
	hipMalloc(&g_my_first_pattern, sizeof(int));

	hipMemcpy(g_seq_length, &seq_length, sizeof(unsigned long), hipMemcpyHostToDevice);
	hipMemcpy(g_pat_number, &my_pat_number, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(g_sequence, sequence, seq_length * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(g_seq_matches, seq_matches, seq_length * sizeof(int), hipMemcpyHostToDevice);
	int init_value = 0;
	hipMemcpy(g_pat_matches, &init_value, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(g_pat_found, my_pat_found, my_pat_number * sizeof(unsigned long), hipMemcpyHostToDevice);

	hipMemcpy(g_my_first_pattern, &my_first_pattern, sizeof(int), hipMemcpyHostToDevice);


	// Ponendo di avere 256 thread per blocco potremmo fare ceil(pat_number/256.0)
	// cosi da calcolare il numero di blocchi necessari per dividere le sequenze da cercare tra i thread
	// Potremmo quindi fare che ogni thread cerca una sola sequenza?

	hipMemcpy(d_pat_length, &pat_length[my_first_pattern], my_pat_number * sizeof(unsigned long), hipMemcpyHostToDevice);

	/*
	Facciamo con 1024 thread per blocco, questo perché il massimo numero di thread per SM nell'architettura Turing è 1024 (max 32 warp per SM, ogni warp è da 32 threads)
	per calcolare il numero di blocchi lo facciamo con ceil(pat_number/1024.0), possiamo pure provare con altre grandezze di blocchi (max numero di blocchi per SM è 16)
	noi stiamo facendo che ogni thread ha una sequenza da vedere, ma se ci sono meno di 1024 sequenze che succede?
	Aggiunto: succede che si genera solo un blocco e lavorano solo tot thread, il resto rimane inutilizzato
	*/

	// Nel caso ci siano più processi MPI che vogliono accedere alla stessa GPU, devo serializzare l'accesso
    // Questa cosa va bene perché tanto ogni rank avrà un insieme di pattern differente dagli altri da cercare
    // Ovviamente tutte le GPU devono essere impegnate contemporaneamente

	MPI_Barrier(MPI_COMM_WORLD);

    // Imposto il device per ogni processo (considerando che i nodi del cluster hanno due GPU ciascuno)
    if (rank % 2 == 0) {
        hipSetDevice(0);
    } else {
        hipSetDevice(1);
    }
    
    // da rivedere
    // rank pari (0) lavora su GPU 0
    if (rank % 2 == 0) {
        sequencer<<<ceil(my_pat_number/1024.0), 1024>>>(g_seq_length, g_pat_number, g_sequence, d_pat_length, d_pattern, g_seq_matches, g_pat_matches, g_pat_found, g_my_first_pattern);
        hipDeviceSynchronize();
    }

    // rank dispari (1) lavora su GPU 1
    if (rank % 2 == 1) {
        sequencer<<<ceil(my_pat_number/1024.0), 1024>>>(g_seq_length, g_pat_number, g_sequence, d_pat_length, d_pattern, g_seq_matches, g_pat_matches, g_pat_found, g_my_first_pattern);
        hipDeviceSynchronize();
    }
    
    MPI_Barrier(MPI_COMM_WORLD);

	// Riporto le variabili, che il kernel ha modificato, utilizzate dal checksum nell'host
	hipMemcpy(seq_matches, g_seq_matches, seq_length * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&pat_matches, g_pat_matches, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(my_pat_found, g_pat_found, my_pat_number * sizeof(unsigned long), hipMemcpyDeviceToHost);

	// Porto i pattern trovati dai rank in un unica struttura (pat_found)
	for( ind=0; ind<my_pat_number; ind++ ) {
		pat_found[ my_first_pattern + ind ] = my_pat_found[ind];
	}

	hipFree(g_seq_matches);
	hipFree(g_pat_matches);
	hipFree(g_pat_found);
	hipFree(d_pattern);
	hipFree(d_pat_length);

	hipDeviceSynchronize();


	// Tutti i rank inviano i propri seq_matches al rank 0
	if(rank > 0){
		MPI_Send(seq_matches, seq_length, MPI_INT, 0, 0, MPI_COMM_WORLD);
	}

    // Il rank 0 si occuperà di sommare i seq_matches di tutti i rank
    if (rank == 0) {
		int *local_seq_matches;
		local_seq_matches = (int *)malloc( sizeof(int) * seq_length );
		
		for (int i = 1; i < size; i++) {
			MPI_Recv(local_seq_matches, seq_length, MPI_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			
			for (unsigned long j = 0; j < seq_length; j++) {
				if (seq_matches[j] >= 0 && local_seq_matches[j] >= 0) {
					seq_matches[j] = seq_matches[j] + local_seq_matches[j] + 1;
				} else {
					if(seq_matches[j] == NOT_FOUND){
						seq_matches[j] = local_seq_matches[j];
					}
					
				}
			}
		}
		
		free(local_seq_matches);
	}
	MPI_Barrier( MPI_COMM_WORLD );

	// Ognuno invia i pattern trovati al rank 0
	//MPI_Gather(pat_found + my_first_pattern, my_patterns, MPI_UNSIGNED_LONG, pat_found, my_patterns, MPI_UNSIGNED_LONG, 0, MPI_COMM_WORLD);
	
	int send_counts[size]; // {133, 133, 134}; // Numero di elementi da inviare per ciascun rank 
	send_counts[rank] = my_pat_number;

	int displs[size]; //= {0, 133, 266};
	displs[rank] = my_first_pattern;
	
	MPI_Allgather(&my_pat_number, 1, MPI_INT, send_counts, 1, MPI_INT, MPI_COMM_WORLD);
	MPI_Allgather(&my_first_pattern, 1, MPI_INT, displs, 1, MPI_INT, MPI_COMM_WORLD);

	MPI_Barrier( MPI_COMM_WORLD );

	MPI_Gatherv(my_pat_found, my_pat_number, MPI_UNSIGNED_LONG, pat_found, send_counts, displs, MPI_UNSIGNED_LONG, 0, MPI_COMM_WORLD);
	
	// Ognuno invia al rank 0 il numero di pattern trovati sommandoli con la reduce
	int total_pat_matches = 0;
	MPI_Reduce(&pat_matches, &total_pat_matches, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);

	// Il rank 0 assegna il valore di total_pat_matches a pat_matches
	if (rank == 0) {
		pat_matches = total_pat_matches;
	}

	/*if (rank == 0) {
		// Debug: Print seq_matches array
		printf("Sequence matches: ");
		for (lind = 0; lind < seq_length; lind++) {
			printf("%d ", seq_matches[lind]);
		}
		printf("\n");
	}*/

	MPI_Barrier( MPI_COMM_WORLD );
	
	/* 7. Check sums */
	unsigned long checksum_matches = 0;
	unsigned long checksum_found = 0;
	if(rank == 0) {
		for( ind=0; ind < pat_number; ind++) {
			if ( pat_found[ind] != (unsigned long)NOT_FOUND )
				checksum_found = ( checksum_found + pat_found[ind] ) % CHECKSUM_MAX;
		}
		for( lind=0; lind < seq_length; lind++) {
			if ( seq_matches[lind] != NOT_FOUND )
				checksum_matches = ( checksum_matches + seq_matches[lind] ) % CHECKSUM_MAX;
		}
	}

	

#ifdef DEBUG
	/* DEBUG: Write results */
	printf("-----------------\n");
	printf("Found start:");
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( " %lu", pat_found[debug_pat] );
	}
	printf("\n");
	printf("-----------------\n");
	printf("Matches:");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( " %d", seq_matches[lind] );
	printf("\n");
	printf("-----------------\n");
#endif // DEBUG

	/* Free local resources */	
	free( sequence );
	free( seq_matches );

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 8. Stop global time */
	MPI_Barrier( MPI_COMM_WORLD );
	ttotal = cp_Wtime() - ttotal;

	/* 9. Output for leaderboard */
	if ( rank == 0 ) {
		printf("\n");
		/* 9.1. Total computation time */
		printf("Time: %lf\n", ttotal );

		/* 9.2. Results: Statistics */
		printf("Result: %d, %lu, %lu\n\n", 
				pat_matches,
				checksum_found,
				checksum_matches );
	}
				
	/* 10. Free resources */	
	int i;
	for( i=0; i<pat_number; i++ ) free( pattern[i] );
	free( pattern );
	free( pat_length );
	free( pat_found );

	/* 11. End */
	MPI_Finalize();
	return 0;
}
